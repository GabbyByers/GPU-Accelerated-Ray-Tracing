#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <math.h>
#include "kernel.cuh"
#include "vec3.cuh"
#include "camera.cuh"
#include "ray.cuh"

__global__ void kernel(camera& camera, Uint8* ptr, unsigned int size, unsigned int width) {
    unsigned int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (i >= size) { return; }
    
    Uint8* RGBA = ptr + (i * 4);
    Uint8& r = *   RGBA;
    Uint8& g = * ++RGBA;
    Uint8& b = * ++RGBA;

    unsigned int x = i % width;
    unsigned int y = i / width;
    double u = x / static_cast<double>(width);
    double v = y / static_cast<double>(size / width);
    u = (2.0 * u) - 1.0;
    v = (2.0 * v) - 1.0;
    v = -v;

    vec3 pixel(u, v, 0.0);
    ray camera_ray(camera.position, camera.direction.add(pixel));
}

void perPixelCalculation(camera& camera, Uint8* scene, unsigned int size, unsigned int width) {
    unsigned int bytes = size * 4;
    Uint8* ptr = nullptr;

    hipMalloc((void**)&ptr, bytes);
    hipMemcpy(ptr, scene, bytes, hipMemcpyHostToDevice);

    unsigned int NUM_THREADS = 1024;
    unsigned int NUM_BLOCKS = (size + NUM_THREADS - 1) / NUM_THREADS;
    kernel <<<NUM_BLOCKS, NUM_THREADS>>> (camera, ptr, size, width);

    hipMemcpy(scene, ptr, bytes, hipMemcpyDeviceToHost);
    hipFree(ptr);
}