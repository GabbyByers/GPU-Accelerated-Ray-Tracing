#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <math.h>
#include "kernel.cuh"
#include "vec3.cuh"
#include "camera.cuh"
#include "ray.cuh"
#include "sphere.cuh"
#include "Enviroment.cuh"

__global__ void kernel(Enviroment enviroment, camera camera, Uint8* gpu_ptr, unsigned int size, unsigned int width) {
    unsigned int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (i >= size) { return; }
    
    Uint8* RGBA = gpu_ptr + (i * 4);
    Uint8& r = *   RGBA;
    Uint8& g = * ++RGBA;
    Uint8& b = * ++RGBA;

    unsigned int x = i % width;
    unsigned int y = i / width;

    float u = x / static_cast<float>(width);
    float v = y / static_cast<float>(size / width);

    u = (2.0f * u) - 1.0f;
    v = (2.0f * v) - 1.0f;

    u = u * (width / static_cast<float>(size / width));
    v = -v;

    vec3 origin = camera.position;
    vec3 base_direction = vec3(u, v, camera.depth);

    vec3 real_direction = base_direction.vectorMatrixMultiplication(camera.rotation);
    
    ray ray(origin, real_direction);
    ray.trace(enviroment.gpu_spheres, enviroment.num_spheres);

    r = ray.r;
    g = ray.g;
    b = ray.b;
}

Uint8* gpuSetup(Uint8* cpu_ptr, unsigned int size) {
    unsigned int bytes = size * 4;
    Uint8* gpu_ptr = nullptr;
    hipMalloc((void**)&gpu_ptr, bytes);
    hipMemcpy(gpu_ptr, cpu_ptr, bytes, hipMemcpyHostToDevice);
    return gpu_ptr;
}

void gpuCalc(Enviroment& enviroment, camera& camera, Uint8* cpu_ptr, Uint8* gpu_ptr, unsigned int size, unsigned int width) {
    unsigned int NUM_THREADS = 512;
    unsigned int NUM_BLOCKS = (size + NUM_THREADS - 1) / NUM_THREADS;
    kernel <<<NUM_BLOCKS, NUM_THREADS>>> (enviroment, camera, gpu_ptr, size, width);

    unsigned int bytes = size * 4;
    hipMemcpy(cpu_ptr, gpu_ptr, bytes, hipMemcpyDeviceToHost);
}

void gpuFree(Uint8* gpu_ptr) {
    hipFree(gpu_ptr);
}