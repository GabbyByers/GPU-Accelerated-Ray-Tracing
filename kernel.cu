#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <math.h>
#include "kernel.cuh"
#include "vec3.cuh"
#include "camera.cuh"
#include "ray.cuh"

__global__ void kernel(camera& camera, Uint8* gpu_ptr, unsigned int size, unsigned int width) {
    unsigned int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (i >= size) { return; }

    Uint8* RGBA = gpu_ptr + (i * 4);
    Uint8& r = *   RGBA;
    Uint8& g = * ++RGBA;
    Uint8& b = * ++RGBA;

    unsigned int x = i % width;
    unsigned int y = i / width;
    double u = x / static_cast<double>(width);
    double v = y / static_cast<double>(size / width);
    u = (2.0 * u) - 1.0;
    v = (2.0 * v) - 1.0;
    v = -v;

    r = abs(u) * 255;
    b = abs(v) * 255;

    //vec3 pixel(u, v, 0.0);
    //ray camera_ray(camera.position, camera.direction.add(pixel));
}

Uint8* gpuSetup(Uint8* cpu_ptr, unsigned int size) {
    unsigned int bytes = size * 4;
    Uint8* gpu_ptr = nullptr;
    hipMalloc((void**)&gpu_ptr, bytes);
    hipMemcpy(gpu_ptr, cpu_ptr, bytes, hipMemcpyHostToDevice);
    return gpu_ptr;
}

void perPixelCalculation(camera& camera, Uint8* cpu_ptr, Uint8* gpu_ptr, unsigned int size, unsigned int width) {
    unsigned int NUM_THREADS = 1024;
    unsigned int NUM_BLOCKS = (size + NUM_THREADS - 1) / NUM_THREADS;
    kernel <<<NUM_BLOCKS, NUM_THREADS>>> (camera, gpu_ptr, size, width);

    unsigned int bytes = size * 4;
    hipMemcpy(cpu_ptr, gpu_ptr, bytes, hipMemcpyDeviceToHost);
    //hipFree(gpu_ptr);
}

void theOldFunction(camera& camera, Uint8* cpu_ptr, unsigned int size, unsigned int width) {
    unsigned int bytes = size * 4;
    Uint8* gpu_ptr = nullptr;

    hipMalloc((void**)&gpu_ptr, bytes);
    hipMemcpy(gpu_ptr, cpu_ptr, bytes, hipMemcpyHostToDevice);

    unsigned int NUM_THREADS = 1024;
    unsigned int NUM_BLOCKS = (size + NUM_THREADS - 1) / NUM_THREADS;
    kernel << <NUM_BLOCKS, NUM_THREADS >> > (camera, gpu_ptr, size, width);

    hipMemcpy(cpu_ptr, gpu_ptr, bytes, hipMemcpyDeviceToHost);
    hipFree(gpu_ptr);
}