#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <math.h>
#include "vec3.cuh"

typedef unsigned char Uint8;

__global__ void kernel(Uint8* ptr, unsigned int size, unsigned int width) {
    unsigned int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (i < size) {
        Uint8* RGBA = ptr + (i * 4);
        Uint8& r = *   RGBA;
        Uint8& g = * ++RGBA;
        Uint8& b = * ++RGBA;

        unsigned int x = i % width;
        unsigned int y = i / width;
        double u = x / static_cast<double>(width);
        double v = y / static_cast<double>(size / width);

        vec3 ray(u, 0.0, v);

        r = ray.x * 255;
        g = ray.y * 255;
        b = ray.z * 255;
    }
}

void perPixelCalculation(Uint8* scene, unsigned int size, unsigned int width) {
    unsigned int bytes = size * 4;
    Uint8* ptr = nullptr;

    hipMalloc((void**)&ptr, bytes);
    hipMemcpy(ptr, scene, bytes, hipMemcpyHostToDevice);

    int NUM_THREADS = 1024;
    int NUM_BLOCKS = (size + NUM_THREADS - 1) / NUM_THREADS;
    kernel <<<NUM_BLOCKS, NUM_THREADS>>> (ptr, size, width);

    hipMemcpy(scene, ptr, bytes, hipMemcpyDeviceToHost);
    hipFree(ptr);
}
