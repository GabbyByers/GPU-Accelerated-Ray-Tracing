#include "Enviroment.cuh"

Enviroment::Enviroment() {}

void Enviroment::addSphere(vec3& position, double radius) {
    num_spheres++;

    // CPU
    Sphere* new_cpu_spheres = new Sphere[num_spheres];
    memcpy(new_cpu_spheres, cpu_spheres, sizeof(Sphere) * (num_spheres - 1));
    delete[] cpu_spheres;
    cpu_spheres = new_cpu_spheres;
    Sphere new_sphere(position, radius);
    cpu_spheres[num_spheres - 1] = new_sphere;

    // GPU
    hipFree(gpu_spheres);
    hipMalloc((void**)&gpu_spheres, sizeof(Sphere) * num_spheres);
    hipMemcpy(gpu_spheres, cpu_spheres, sizeof(Sphere) * num_spheres, hipMemcpyHostToDevice);
}

void Enviroment::destroy() {
    delete[] cpu_spheres;
    hipFree(gpu_spheres);
}