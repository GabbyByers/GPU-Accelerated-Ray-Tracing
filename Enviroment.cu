#include "Enviroment.cuh"
#include <string.h>

Enviroment::Enviroment() {}

void Enviroment::addSphere(Sphere sphere) {
    num_spheres++;

    // CPU
    Sphere* new_cpu_spheres = new Sphere[num_spheres];
    memcpy(new_cpu_spheres, cpu_spheres, sizeof(Sphere) * (num_spheres - 1));
    delete[] cpu_spheres;
    cpu_spheres = new_cpu_spheres;
    cpu_spheres[num_spheres - 1] = sphere;

    // GPU
    hipFree(gpu_spheres);
    hipMalloc((void**)&gpu_spheres, sizeof(Sphere) * num_spheres);
    hipMemcpy(gpu_spheres, cpu_spheres, sizeof(Sphere) * num_spheres, hipMemcpyHostToDevice);
}

void Enviroment::destroy() {
    delete[] cpu_spheres;
    hipFree(gpu_spheres);
}

void Enviroment::updateSpheres(){
    hipMemcpy(gpu_spheres, cpu_spheres, sizeof(Sphere) * num_spheres, hipMemcpyHostToDevice);
}