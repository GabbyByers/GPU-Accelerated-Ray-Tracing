#include "sphere.cuh"

Sphere::Sphere() {}

Sphere::Sphere(vec3 _position, float _radius, Uint8 _r, Uint8 _g, Uint8 _b) {
	position = _position;
	radius = _radius;
	r = _r;
	g = _g;
	b = _b;
}