#include "vec3.cuh"

__host__ __device__ vec3::vec3() {}

__host__ __device__ vec3::vec3(float _x, float _y) {
    x = _x;
    y = _y;
}

__host__ __device__ vec3::vec3(float _x, float _y, float _z) {
    x = _x;
    y = _y;
    z = _z;
}

__host__ __device__ vec3 vec3::add(vec3 vect) {
    vec3 result;
    result.x = x + vect.x;
    result.y = y + vect.y;
    result.z = z + vect.z;
    return result;
}

__host__ __device__ vec3 vec3::vectorMatrixMultiplication(matrix roation) {
    vec3 result;
    // myself multiplied by the matrix
    return result;
}