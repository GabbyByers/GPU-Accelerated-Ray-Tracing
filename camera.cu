#include "camera.cuh"
#include <SFML\Graphics.hpp>

camera::camera() {}

void camera::control(float speed) {
    if (sf::Keyboard::isKeyPressed(sf::Keyboard::W))
    { 
        position.z += speed;// = position.add(vec3(0, 0, speed));
    }
    if (sf::Keyboard::isKeyPressed(sf::Keyboard::A))
    {
        position.x += -speed;// = position.add(vec3(-speed, 0, 0));
    }
    if (sf::Keyboard::isKeyPressed(sf::Keyboard::S))
    {
        position.z += -speed;// = position.add(vec3(0, 0, -speed));
    }
    if (sf::Keyboard::isKeyPressed(sf::Keyboard::D))
    {
        position.x += speed;// = position.add(vec3(speed, 0, 0));
    }
    if (sf::Keyboard::isKeyPressed(sf::Keyboard::Space))
    {
        position.y += speed;// = position.add(vec3(0, speed, 0));
    }
    if (sf::Keyboard::isKeyPressed(sf::Keyboard::LShift))
    {
        position.y += -speed;// = position.add(vec3(0, -speed, 0));
    }
}