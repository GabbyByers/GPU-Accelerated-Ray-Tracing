#include "camera.cuh"

__host__ __device__ camera::camera() {}

__host__ __device__ camera::camera(vec3& _position, vec3& _direction) {
	position = _position;
	direction = _direction;
}