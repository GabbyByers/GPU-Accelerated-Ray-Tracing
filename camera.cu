#include "camera.cuh"
#include <SFML\Graphics.hpp>

camera::camera() {
    real_direction = base_direction;
}

void camera::calculateRotationMatrix() {}

void camera::control(float speed) {
    if (sf::Keyboard::isKeyPressed(sf::Keyboard::W))
    { 
        position.z += speed;
    }
    if (sf::Keyboard::isKeyPressed(sf::Keyboard::S))
    {
        position.z += -speed;
    }
    if (sf::Keyboard::isKeyPressed(sf::Keyboard::A))
    {
        position.x += -speed;
    }
    if (sf::Keyboard::isKeyPressed(sf::Keyboard::D))
    {
        position.x += speed;
    }
    if (sf::Keyboard::isKeyPressed(sf::Keyboard::Space))
    {
        position.y += speed;
    }
    if (sf::Keyboard::isKeyPressed(sf::Keyboard::LShift))
    {
        position.y += -speed;
    }
}