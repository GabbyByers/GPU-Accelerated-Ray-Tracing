#include "hip/hip_runtime.h"
#include "ray.cuh"

__device__ ray::ray() {}

__device__ ray::ray(vec3 _origin, vec3 _direction) {
    origin = _origin;
    direction = _direction;
}

__device__ void ray::trace(const Sphere* gpu_spheres, unsigned int size) {
    
    float dy = direction.y - origin.y;
    float dxz = sqrt((direction.x - origin.x) * (direction.x - origin.x) + (direction.z - origin.z) * (direction.z - origin.z));
    float slope = dy / dxz;
    r = 0;
    g = 150 - slope * 100;
    b = 255;

    float current_t = FLT_MAX;
    for (int i = 0; i < size; i++) {
        const Sphere& sphere = gpu_spheres[i];
        float new_t = intersectSphere(sphere);
        if (new_t < current_t) {
            current_t = new_t;
            r = sphere.r;
            g = sphere.g;
            b = sphere.b;
        }
    }
}

__device__ float ray::intersectSphere(const Sphere& sphere) {
    vec3 A = origin;
    vec3 B = direction;
    vec3 C = sphere.position;

    float r = sphere.radius;

    float a = (B.x * B.x) + (B.y * B.y) + (B.z * B.z);
    float b = (2.0f * A.x * B.x) + (-2.0f * B.x * C.x) + (2.0f * A.y * B.y) + (-2.0f * B.y * C.y) + (2.0f * A.z * B.z) + (-2.0f * B.z * C.z);
    float c = (A.x * A.x) + (-2.0f * A.x * C.x) + (C.x * C.x) + (A.y * A.y) + (-2.0f * A.y * C.y) + (C.y * C.y) + (A.z * A.z) + (-2.0f * A.z * C.z) + (C.z * C.z) - (r * r);
    
    float discriminant = (b * b) + (-4.0f * a * c);
    if (discriminant <= 0) { return FLT_MAX; }
 
    float t0 = (-b + sqrt(discriminant)) / (2.0f * a);
    float t1 = (-b - sqrt(discriminant)) / (2.0f * a);
  
    if (t0 <= 0.0f) { t0 = FLT_MAX; }
    if (t1 <= 0.0f) { t1 = FLT_MAX; }
  
    if (t0 <= t1) { return t0; }
    if (t1 <= t0) { return t1; }
}