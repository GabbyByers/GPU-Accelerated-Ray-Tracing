#include "hip/hip_runtime.h"
#include "ray.cuh"

__device__ ray::ray() {}

__device__ ray::ray(vec3 _origin, vec3 _direction) {
    origin = _origin;
    direction = _direction;
}

__device__ void ray::trace(const Sphere* gpu_spheres, unsigned int size) {
    
    // color if no collision
    double dy = direction.y - origin.y;
    double dxz = sqrt((direction.x - origin.x) * (direction.x - origin.x) + (direction.z - origin.z) * (direction.z - origin.z));
    double slope = dy / dxz;
    r = 0;
    g = 150 - slope * 100;
    b = 255;

    for (int i = 0; i < size; i++) {
        Sphere sphere = gpu_spheres[i];
        // check collision w/ spheres, over-write color if there is a collision.
    }
}