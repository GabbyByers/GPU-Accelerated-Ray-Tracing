#include "matrix.cuh"

#include <stdio.h>
#include <stdlib.h>

matrix::matrix() {

	cpu_matrix = new float*[3];
	for (int i = 0; i < 3; i++) {
		cpu_matrix[i] = new float[3];
	}

	hipMalloc((void**)&gpu_matrix, sizeof(float*) * 3);
	for (int i = 0; i < 3; i++) {
		hipMalloc((void**)&gpu_matrix[i], sizeof(float) * 3);
	}

	//update();
}

void matrix::update() {
	for (int i = 0; i < 3; i++) {
		hipMemcpy(gpu_matrix[i], cpu_matrix[i], sizeof(float) * 3, hipMemcpyHostToDevice);
	}
}



