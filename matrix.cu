#include "matrix.cuh"

#include <stdio.h>
#include <stdlib.h>

matrix::matrix()
{	
	cpu_matrix = new float[9];
	for (int i = 0; i < 9; i++) { cpu_matrix[i] = 0.0f; }
	
	hipMalloc((void**)&gpu_matrix, sizeof(float) * 9);
	
	update();
}

void matrix::update()
{
	hipMemcpy(gpu_matrix, cpu_matrix, sizeof(float) * 9, hipMemcpyHostToDevice);
}